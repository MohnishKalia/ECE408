#include "hip/hip_runtime.h"

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// helper for computing malloc size
#define DATA_SIZE(inputLen) (inputLen * sizeof(float))
// map 2d to 1d array
#define IDX_2D(x, y, stride) (y * stride + x)
// thread block size for tiling
#define BLOCK_WIDTH 32

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
  __shared__ float stA[BLOCK_WIDTH][BLOCK_WIDTH];
  __shared__ float stB[BLOCK_WIDTH][BLOCK_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int col = bx * blockDim.x + tx;
  int row = by * blockDim.y + ty;

  int WIDTH = numAColumns; // or numBRows

  // over q tiles, load and compute
  float result = 0;
  for (int q = 0; q < ceil((1.0 * WIDTH) / BLOCK_WIDTH); q++) {
    // Phase 1: load from source matricies into shared mem constructs
    int offsetA = q * BLOCK_WIDTH + tx; // which col
    int offsetB = q * BLOCK_WIDTH + ty; // which row
    int threadTargetIdxA = row * numAColumns + offsetA;
    int threadTargetIdxB = offsetB * numBColumns + col;

    // handle halo cells
    if (row < numCRows && (q*BLOCK_WIDTH+tx) < WIDTH)
      stA[ty][tx] = A[threadTargetIdxA];
    else {
      //printf("Kernel A halo at thread %dx%d\n", tx, ty);
      stA[ty][tx] = 0;
    }

    // handle halo cells
    if ((q*BLOCK_WIDTH+ty) < WIDTH && col < numCColumns)
      stB[ty][tx] = B[threadTargetIdxB];
    else {
      //printf("Kernel B halo at thread %dx%d\n", tx, ty);
      stB[ty][tx] = 0;
    }
    __syncthreads();
    
    // Phase 2: compute matrix mul with smaller subunit matricies
    if (col < numCColumns && row < numCRows) {
      for (int i = 0; i < BLOCK_WIDTH; i++) {
        result += stA[ty][i] * stB[i][tx];
      }
    }
    __syncthreads();
  }
  
  if (col < numCColumns && row < numCRows) {
    C[IDX_2D(col, row, numCColumns)] = result;
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  if (numAColumns != numBRows) {
    wbLog(ERROR, "Invalid A and B dimensions for matrix multiplication.");
    return 1;
  }
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  int cInputSize = numCRows * numCColumns;
  hostC = (float *)malloc(DATA_SIZE(cInputSize));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
  wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  int aInputSize = numARows * numAColumns;
  int bInputSize = numBRows * numBColumns;
  wbCheck(hipMalloc((void **)&deviceA, DATA_SIZE(aInputSize)));
  wbCheck(hipMalloc((void **)&deviceB, DATA_SIZE(bInputSize)));
  wbCheck(hipMalloc((void **)&deviceC, DATA_SIZE(cInputSize)));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  wbCheck(hipMemcpy(deviceA, hostA, DATA_SIZE(aInputSize), hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceB, hostB, DATA_SIZE(bInputSize), hipMemcpyHostToDevice));

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(
    ceil((1.0*numCColumns)/BLOCK_WIDTH),
    ceil((1.0*numCRows)/BLOCK_WIDTH),
    1
  );
  dim3 DimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);
  wbLog(TRACE, "The dimensions of DimGrid are ", DimGrid.x, " x ", DimGrid.y, " x ", DimGrid.z);
  wbLog(TRACE, "The dimensions of DimBlock are ", DimBlock.x, " x ", DimBlock.y, " x ", DimBlock.z);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, 
                                              numARows, numAColumns, 
                                              numBRows, numBColumns,
                                              numCRows, numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  wbCheck(hipMemcpy(hostC, deviceC, DATA_SIZE(cInputSize), hipMemcpyDeviceToHost));

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  wbCheck(hipFree(deviceA));
  wbCheck(hipFree(deviceB));
  wbCheck(hipFree(deviceC));

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
