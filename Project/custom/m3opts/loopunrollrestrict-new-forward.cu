#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16

__global__ void conv_forward_kernel(float * __restrict__ output, const float * __restrict__ input, const float * __restrict__ mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here

    // same as grid setup
    int W_size = ceil(1.0f*W_out/TILE_WIDTH); // number of horizontal tiles per output map
    int H_size = ceil(1.0f*H_out/TILE_WIDTH); // number of vertical tiles per output map
    int b = blockIdx.z;
    int m = blockIdx.x;
    int h = (blockIdx.y / W_size) * TILE_WIDTH + threadIdx.y; // target h of output
    int w = (blockIdx.y % W_size) * TILE_WIDTH + threadIdx.x; // target w of output

    // each thread ran should be within output bounds, otherwise return
    if (w < 0 || w >= W_out || h < 0 || h >= H_out)
        return;

    float acc = 0.0f;

    // loop unroll logic based on mask size
    if (K <= 3) {
        // low K, just do normally, no real performance gain from unroll
        for (int c = 0; c < C; c++) { // sum over all input channels
            for (int p = 0; p < K; p++) { // loop over KxK filter
                for (int q = 0; q < K; q++) {
                    int h_idx = (h * S + p);
                    int w_idx = (w * S + q);
                    // if target idx is not within input bounds, use 0, otherwise grab value
                    if (!(w_idx < 0 || w_idx >= W || h_idx < 0 || h_idx >= H))
                        acc += in_4d(b, c, h_idx, w_idx) * mask_4d(m, c, p, q);
                }
            }
        }
    } 
    else if (K <= 7)
    {
        #define Q_UNROLL 4
        // middling K value, get a few unroll ops
        for (int c = 0; c < C; c++) { // sum over all input channels
            for (int p = 0; p < K; p++) { // loop over KxK filter
                // turn q into # of loop iters to target instead of just q
                int qbnd = ceil(1.0f * K / Q_UNROLL);
                for (int q = 0; q < qbnd; q++) {
                    // calc q0..3 and w0..3
                    int h_idx = (h * S + p);
                    int q0 = q * Q_UNROLL + 0;
                    int q1 = q * Q_UNROLL + 1;
                    int q2 = q * Q_UNROLL + 2;
                    int q3 = q * Q_UNROLL + 3;
                    int w0_idx = (w * S + q0);
                    int w1_idx = (w * S + q1);
                    int w2_idx = (w * S + q2);
                    int w3_idx = (w * S + q3);
                    // if target idx is not within input bounds, or q# is too far, use 0, otherwise grab value
                    if (!(w0_idx < 0 || w0_idx >= W || h_idx < 0 || h_idx >= H) && q0 < K)
                        acc += in_4d(b, c, h_idx, w0_idx) * mask_4d(m, c, p, q0);
                    if (!(w1_idx < 0 || w1_idx >= W || h_idx < 0 || h_idx >= H) && q1 < K)
                        acc += in_4d(b, c, h_idx, w1_idx) * mask_4d(m, c, p, q1);
                    if (!(w2_idx < 0 || w2_idx >= W || h_idx < 0 || h_idx >= H) && q2 < K)
                        acc += in_4d(b, c, h_idx, w2_idx) * mask_4d(m, c, p, q2);
                    if (!(w3_idx < 0 || w3_idx >= W || h_idx < 0 || h_idx >= H) && q3 < K)
                        acc += in_4d(b, c, h_idx, w3_idx) * mask_4d(m, c, p, q3);
                }
            }
        }
        #undef Q_UNROLL
    }
    else
    {
        #define Q_UNROLL 8
        // beyond this K >= 8, performance gains fetter out
        // only a few unrolls
        for (int c = 0; c < C; c++) { // sum over all input channels
            for (int p = 0; p < K; p++) { // loop over KxK filter
                // turn q into # of loop iters to target instead of just q
                int qbnd = ceil(1.0f * K / Q_UNROLL);
                for (int q = 0; q < qbnd; q++) {
                    // calc q0..7 and w0..7
                    int h_idx = (h * S + p);
                    int q0 = q * Q_UNROLL + 0;
                    int q1 = q * Q_UNROLL + 1;
                    int q2 = q * Q_UNROLL + 2;
                    int q3 = q * Q_UNROLL + 3;
                    int q4 = q * Q_UNROLL + 4;
                    int q5 = q * Q_UNROLL + 5;
                    int q6 = q * Q_UNROLL + 6;
                    int q7 = q * Q_UNROLL + 7;
                    int w0_idx = (w * S + q0);
                    int w1_idx = (w * S + q1);
                    int w2_idx = (w * S + q2);
                    int w3_idx = (w * S + q3);
                    int w4_idx = (w * S + q4);
                    int w5_idx = (w * S + q5);
                    int w6_idx = (w * S + q6);
                    int w7_idx = (w * S + q7);
                    // if target idx is not within input bounds, or q# is too far, use 0, otherwise grab value
                    if (!(w0_idx < 0 || w0_idx >= W || h_idx < 0 || h_idx >= H) && q0 < K)
                        acc += in_4d(b, c, h_idx, w0_idx) * mask_4d(m, c, p, q0);
                    if (!(w1_idx < 0 || w1_idx >= W || h_idx < 0 || h_idx >= H) && q1 < K)
                        acc += in_4d(b, c, h_idx, w1_idx) * mask_4d(m, c, p, q1);
                    if (!(w2_idx < 0 || w2_idx >= W || h_idx < 0 || h_idx >= H) && q2 < K)
                        acc += in_4d(b, c, h_idx, w2_idx) * mask_4d(m, c, p, q2);
                    if (!(w3_idx < 0 || w3_idx >= W || h_idx < 0 || h_idx >= H) && q3 < K)
                        acc += in_4d(b, c, h_idx, w3_idx) * mask_4d(m, c, p, q3);
                    if (!(w4_idx < 0 || w4_idx >= W || h_idx < 0 || h_idx >= H) && q4 < K)
                        acc += in_4d(b, c, h_idx, w4_idx) * mask_4d(m, c, p, q4);
                    if (!(w5_idx < 0 || w5_idx >= W || h_idx < 0 || h_idx >= H) && q5 < K)
                        acc += in_4d(b, c, h_idx, w5_idx) * mask_4d(m, c, p, q5);
                    if (!(w6_idx < 0 || w6_idx >= W || h_idx < 0 || h_idx >= H) && q6 < K)
                        acc += in_4d(b, c, h_idx, w6_idx) * mask_4d(m, c, p, q6);
                    if (!(w7_idx < 0 || w7_idx >= W || h_idx < 0 || h_idx >= H) && q7 < K)
                        acc += in_4d(b, c, h_idx, w7_idx) * mask_4d(m, c, p, q7);
                }
            }
        }
        #undef Q_UNROLL
    }
    

    // after accumulating, set to output value
    out_4d(b, m, h, w) = acc;

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    #define wbCheck(stmt)                                                     \
    do {                                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            std::cout<<"Failed to run stmt: "<<#stmt<<std::endl;    \
            std::cout<<"CUDA error: "<<hipGetErrorString(err)<<std::endl;    \
            exit(-1);                                                         \
        }                                                                     \
    } while (0)

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    size_t dop_sz = B * M * H_out * W_out * sizeof(float);
    size_t dip_sz = B * C * H * W * sizeof(float);
    size_t dmp_sz = M * C * K * K * sizeof(float);
    wbCheck(hipMalloc((void **)device_output_ptr, dop_sz));
    wbCheck(hipMalloc((void **)device_input_ptr, dip_sz));
    wbCheck(hipMalloc((void **)device_mask_ptr, dmp_sz));

    // wbCheck(hipMemcpy(*device_output_ptr, host_output, dop_sz, hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(*device_input_ptr, host_input, dip_sz, hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(*device_mask_ptr, host_mask, dmp_sz, hipMemcpyHostToDevice));

    #undef wbCheck
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel

    // same as inside kernel
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int W_size = ceil(1.0f*W_out/TILE_WIDTH); // number of horizontal tiles per output map
    int H_size = ceil(1.0f*H_out/TILE_WIDTH); // number of vertical tiles per output map
    int tileNums = H_size * W_size; // total number of tiles per map
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1); // output tile for untiled code
    dim3 DimGrid(M, tileNums, B);
    std::cout<<"DimBlock: "<<DimBlock.x<<"x"<<DimBlock.y<<"x"<<DimBlock.z<<std::endl;
    std::cout<<"DimGrid: "<<DimGrid.x<<"x"<<DimGrid.y<<"x"<<DimGrid.z<<std::endl;
    conv_forward_kernel<<<DimGrid, DimBlock>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    hipDeviceSynchronize();
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{

    #define wbCheck(stmt)                                                     \
    do {                                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            std::cout<<"Failed to run stmt: "<<#stmt<<std::endl;    \
            std::cout<<"CUDA error: "<<hipGetErrorString(err)<<std::endl;    \
            exit(-1);                                                         \
        }                                                                     \
    } while (0)

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    size_t dop_sz = B * M * H_out * W_out * sizeof(float);

    // Copy the output back to host
    wbCheck(hipMemcpy(host_output, device_output, dop_sz, hipMemcpyDeviceToHost));

   
    // Free device memory
    wbCheck(hipFree(device_input));
    wbCheck(hipFree(device_output));
    wbCheck(hipFree(device_mask));

    #undef wbCheck
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
