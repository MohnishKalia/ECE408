#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 18

/*
Op Time: 13.7135 ms
Op Time: 30.2494 ms
~44.5ms
*/

__global__ void conv_forward_kernel(float * __restrict__ output, const float * __restrict__ input, const float * __restrict__ mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here

    // same as grid setup
    const int W_size = ceil(1.0f*W_out/TILE_WIDTH); // number of horizontal tiles per output map
    const int H_size = ceil(1.0f*H_out/TILE_WIDTH); // number of vertical tiles per output map
    const int b = blockIdx.z;
    const int m = blockIdx.x;
    const int h = (blockIdx.y / W_size) * TILE_WIDTH + threadIdx.y; // target h of output
    const int w = (blockIdx.y % W_size) * TILE_WIDTH + threadIdx.x; // target w of output

    // each thread ran should be within output bounds, otherwise return
    if (w < 0 || w >= W_out || h < 0 || h >= H_out)
        return;

    float acc = 0.0f;

    // loop unroll logic based on mask size
    if (K <= 3) {
        // low K, just do normally, no real performance gain from unroll
        for (int c = 0; c < C; c++) { // sum over all input channels
            for (int p = 0; p < K; p++) { // loop over KxK filter
                for (int q = 0; q < K; q++) {
                    int h_idx = (h * S + p);
                    int w_idx = (w * S + q);
                    // if target idx is not within input bounds, use 0, otherwise grab value
                    if (!(w_idx < 0 || w_idx >= W || h_idx < 0 || h_idx >= H))
                        acc += in_4d(b, c, h_idx, w_idx) * mask_4d(m, c, p, q);
                }
            }
        }
    } 
    else if (K <= 7)
    {
        // 3 is best in this K range from profiling included in report
        #define Q_UNROLL 3
        // turn q into # of loop iters to target instead of just q
        const int qbnd = ceil(1.0f * K / Q_UNROLL);
        // middling K value, get a few unroll ops
        for (int c = 0; c < C; c++) { // sum over all input channels
            for (int p = 0; p < K; p++) { // loop over KxK filter
                // if our height out of bounds, dont bother
                int h_idx = (h * S + p);
                if (!(h_idx < 0 || h_idx >= H)) {
                    for (int q = 0; q < qbnd; q++) {
                        // calc q0..2 and w0..2
                        int q0 = q * Q_UNROLL + 0;
                        int q1 = q * Q_UNROLL + 1;
                        int q2 = q * Q_UNROLL + 2;
                        // int q3 = q * Q_UNROLL + 3;
                        int w0_idx = (w * S + q0);
                        int w1_idx = (w * S + q1);
                        int w2_idx = (w * S + q2);
                        // int w3_idx = (w * S + q3);
                        // if target idx is not within input bounds, or q# is too far, use 0, otherwise grab value
                        if (!(w0_idx < 0 || w0_idx >= W) && q0 < K)
                            acc += in_4d(b, c, h_idx, w0_idx) * mask_4d(m, c, p, q0);
                        if (!(w1_idx < 0 || w1_idx >= W) && q1 < K)
                            acc += in_4d(b, c, h_idx, w1_idx) * mask_4d(m, c, p, q1);
                        if (!(w2_idx < 0 || w2_idx >= W) && q2 < K)
                            acc += in_4d(b, c, h_idx, w2_idx) * mask_4d(m, c, p, q2);
                        // if (!(w3_idx < 0 || w3_idx >= W) && q3 < K)
                        //     acc += in_4d(b, c, h_idx, w3_idx) * mask_4d(m, c, p, q3);
                    }
                }
            }
        }
        #undef Q_UNROLL
    }
    else
    {
        // 8 should scale to larger Ks much better, as well as being at least one full iteration
        #define Q_UNROLL 8
        // beyond this K >= 8, performance gains fetter out
        // only a few unrolls
        for (int c = 0; c < C; c++) { // sum over all input channels
            for (int p = 0; p < K; p++) { // loop over KxK filter
                // if our height out of bounds, dont bother
                int h_idx = (h * S + p);
                if (!(h_idx < 0 || h_idx >= H)) {
                    // turn q into # of loop iters to target instead of just q
                    int qbnd = ceil(1.0f * K / Q_UNROLL);
                    for (int q = 0; q < qbnd; q++) {
                        // calc q0..7 and w0..7
                        int h_idx = (h * S + p);
                        int q0 = q * Q_UNROLL + 0;
                        int q1 = q * Q_UNROLL + 1;
                        int q2 = q * Q_UNROLL + 2;
                        int q3 = q * Q_UNROLL + 3;
                        int q4 = q * Q_UNROLL + 4;
                        int q5 = q * Q_UNROLL + 5;
                        int q6 = q * Q_UNROLL + 6;
                        int q7 = q * Q_UNROLL + 7;
                        int w0_idx = (w * S + q0);
                        int w1_idx = (w * S + q1);
                        int w2_idx = (w * S + q2);
                        int w3_idx = (w * S + q3);
                        int w4_idx = (w * S + q4);
                        int w5_idx = (w * S + q5);
                        int w6_idx = (w * S + q6);
                        int w7_idx = (w * S + q7);
                        // if target idx is not within input bounds, or q# is too far, use 0, otherwise grab value
                        if (!(w0_idx < 0 || w0_idx >= W) && q0 < K)
                            acc += in_4d(b, c, h_idx, w0_idx) * mask_4d(m, c, p, q0);
                        if (!(w1_idx < 0 || w1_idx >= W) && q1 < K)
                            acc += in_4d(b, c, h_idx, w1_idx) * mask_4d(m, c, p, q1);
                        if (!(w2_idx < 0 || w2_idx >= W) && q2 < K)
                            acc += in_4d(b, c, h_idx, w2_idx) * mask_4d(m, c, p, q2);
                        if (!(w3_idx < 0 || w3_idx >= W) && q3 < K)
                            acc += in_4d(b, c, h_idx, w3_idx) * mask_4d(m, c, p, q3);
                        if (!(w4_idx < 0 || w4_idx >= W) && q4 < K)
                            acc += in_4d(b, c, h_idx, w4_idx) * mask_4d(m, c, p, q4);
                        if (!(w5_idx < 0 || w5_idx >= W) && q5 < K)
                            acc += in_4d(b, c, h_idx, w5_idx) * mask_4d(m, c, p, q5);
                        if (!(w6_idx < 0 || w6_idx >= W) && q6 < K)
                            acc += in_4d(b, c, h_idx, w6_idx) * mask_4d(m, c, p, q6);
                        if (!(w7_idx < 0 || w7_idx >= W) && q7 < K)
                            acc += in_4d(b, c, h_idx, w7_idx) * mask_4d(m, c, p, q7);
                    }
                }
            }
        }
        #undef Q_UNROLL
    }
    

    // after accumulating, set to output value
    out_4d(b, m, h, w) = acc;

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    #define wbCheck(stmt)                                                     \
    do {                                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            std::cout<<"Failed to run stmt: "<<#stmt<<std::endl;    \
            std::cout<<"CUDA error: "<<hipGetErrorString(err)<<std::endl;    \
            exit(-1);                                                         \
        }                                                                     \
    } while (0)

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    size_t dop_sz = B * M * H_out * W_out * sizeof(float);
    size_t dip_sz = B * C * H * W * sizeof(float);
    size_t dmp_sz = M * C * K * K * sizeof(float);
    wbCheck(hipMalloc((void **)device_output_ptr, dop_sz));
    wbCheck(hipMalloc((void **)device_input_ptr, dip_sz));
    wbCheck(hipMalloc((void **)device_mask_ptr, dmp_sz));

    // wbCheck(hipMemcpy(*device_output_ptr, host_output, dop_sz, hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(*device_input_ptr, host_input, dip_sz, hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(*device_mask_ptr, host_mask, dmp_sz, hipMemcpyHostToDevice));

    #undef wbCheck
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel

    // same as inside kernel
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    const int W_size = ceil(1.0f*W_out/TILE_WIDTH); // number of horizontal tiles per output map
    const int H_size = ceil(1.0f*H_out/TILE_WIDTH); // number of vertical tiles per output map
    const int tileNums = H_size * W_size; // total number of tiles per map
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1); // output tile for untiled code
    dim3 DimGrid(M, tileNums, B);
    std::cout<<"DimBlock: "<<DimBlock.x<<"x"<<DimBlock.y<<"x"<<DimBlock.z<<std::endl;
    std::cout<<"DimGrid: "<<DimGrid.x<<"x"<<DimGrid.y<<"x"<<DimGrid.z<<std::endl;
    conv_forward_kernel<<<DimGrid, DimBlock>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    hipDeviceSynchronize();
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{

    #define wbCheck(stmt)                                                     \
    do {                                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            std::cout<<"Failed to run stmt: "<<#stmt<<std::endl;    \
            std::cout<<"CUDA error: "<<hipGetErrorString(err)<<std::endl;    \
            exit(-1);                                                         \
        }                                                                     \
    } while (0)

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    size_t dop_sz = B * M * H_out * W_out * sizeof(float);

    // Copy the output back to host
    wbCheck(hipMemcpy(host_output, device_output, dop_sz, hipMemcpyDeviceToHost));

   
    // Free device memory
    wbCheck(hipFree(device_input));
    wbCheck(hipFree(device_output));
    wbCheck(hipFree(device_mask));

    #undef wbCheck
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
